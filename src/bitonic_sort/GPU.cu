#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

extern "C" {
    #include "GPU.h"
}

/*
* bitonic_kernel
* bitonic sort comparison implemented in CUDA
*/
__global__ void bitonic_kernel(int *vect, int step, int jump){
    int itm = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iA = (itm / jump) * jump * 2 + (itm % jump);
    int iB = iA + jump;
    unsigned char asc = ((((iA / (step * 2))) % 2) == 0);
    if((asc && (vect[iA] > vect[iB]) || (!asc && (vect[iA] < vect[iB])))){
        int tmp = vect[iA];
        vect[iA] = vect[iB];
        vect[iB] = tmp;
    }
}

extern "C" {

    int *vect, *cuda_vect, num_items;

    /*
    * init_cuda
    * Init cuda variables and data structures.
    */
    int *init_cuda(int size){
        num_items = size;
        #ifdef __CUDA_SHARED_MEM__
            // create vector in GPU common memory
            hipMallocManaged(&cuda_vect, size * sizeof(int));
            return cuda_vect;
        #else
            // create vector in GPU memory and RAM memory
            hipMalloc(&cuda_vect, sizeof(int) * size);
            vect = (int *)malloc(sizeof(int)*size);
            return vect;
        #endif
    }

    /*
    * end_cuda
    * Deinit cuda variables and data structures.
    */
    void end_cuda(){
        hipFree(cuda_vect);
        #ifndef __CUDA_SHARED_MEM__
            free(vect);
        #endif
        // reset cuda device
        hipDeviceReset();
    }

    /*
    * bitonic_sort
    * bitonic sort based in GPU. Front GPU function
    */
    void bitonic_sort(){
        int num_blocks = (num_items / THREADS_PER_BLOCK) / 2;
        #ifndef __CUDA_SHARED_MEM__
            // Copy numbers to GPU memory
            hipMemcpy(cuda_vect, vect, sizeof(int) * num_items, hipMemcpyHostToDevice);
        #endif
        // Bitonic sort steps an jumps
        for(int step = 1; step < (num_items); step *= 2){
            for(int j = step; j > 0; j /= 2){
                bitonic_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(cuda_vect, step, j);
                hipDeviceSynchronize();
            }
        }
        #ifndef __CUDA_SHARED_MEM__
            // Copy result to main memory
            hipMemcpy(vect, cuda_vect, sizeof(int)*num_items, hipMemcpyDeviceToHost);
        #endif
    }
}
